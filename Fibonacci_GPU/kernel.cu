#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include "Fibonacci.h"
#include <stdio.h>
#include <memory>
#include <math.h>


int initial_fibonacci_run(int fib, int currentDepth, int targetDepth);
__global__ void CUDA_Fibonacci(int *fib, int *result);
__device__ void recursive_fibonacci(int fib, int *result);

// Method to get the number of Stream Processors on the current device
int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
    case 2: // Fermi
        if (devProp.minor == 1) cores = mp * 48;
        else cores = mp * 32;
        break;
    case 3: // Kepler
        cores = mp * 192;
        break;
    case 5: // Maxwell
        cores = mp * 128;
        break;
    case 6: // Pascal
        if (devProp.minor == 1) cores = mp * 128;
        else if (devProp.minor == 0) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    default:
        printf("Unknown device type\n");
        break;
    }
    return cores;
}

// Perform the Recursive solution to the Fibonacci Sequence
// Note: This assumes only one graphics card is installed
// Always grabs the first card
int calc_CUDA_Fibonacci(int number){
    //get number of cores

    // The number of Graphics cards in this computer
    int deviceCount = 0;
    // The Index Device we're going to use (Default 0)
    int currentDevice = 0;
    // The number of CUDA cores on the device being used
    int CUDACoreCount = 0;
    // How deep we need to go into recursion to get the optimal number of threads.
    int depth = 0;
    // Error logging
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0){
        printf("Cannot find any CUDA device.");
        exit(EXIT_FAILURE);
    }

    // Get the information on the current device
    hipSetDevice(currentDevice);
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, currentDevice);

    // Get the number of CUDA Cores on the current device
    CUDACoreCount = getSPcores(deviceProperties);

    // How deep we need to go into the recursion before we spawn threads
    depth = floor(log2((double)CUDACoreCount));

    return initial_fibonacci_run(number, 0, depth);


}

// Method for drilling down to the right level to spawn the correct number of GPU threads
int initial_fibonacci_run(int fib, int currentDepth, int targetDepth){
    if (fib <= 1)
        return fib;

    if (currentDepth < targetDepth)
        return initial_fibonacci_run(fib - 1, currentDepth++, targetDepth) + initial_fibonacci_run(fib - 2, currentDepth++, targetDepth);

    int *d_fib, *d_result;
    int size = sizeof(int);

    hipMalloc((void**)&d_fib, size);
    hipMalloc((void**)&d_result, size);

    hipMemcpy(d_fib, &fib, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, 0, size, hipMemcpyHostToDevice);


    CUDA_Fibonacci<<<1, 1>>>(d_fib, d_result);
    int result = 0;
    hipMemcpy(&result, d_result, size, hipMemcpyDeviceToHost);
    

    hipFree(d_fib);
    hipFree(d_result);
    return result;
}

// Call to the device to start being recursive on a thread
__global__ void CUDA_Fibonacci(int *fib, int *result){
    recursive_fibonacci(*fib, result);
}

// Recursive Fibonacci to run on the GPU Thread
__device__ void recursive_fibonacci(int fib, int *result){
    if (fib <= 1){
        *result += fib;
        return;
    }
    recursive_fibonacci(fib - 1, result);
    recursive_fibonacci(fib - 2, result);
}
